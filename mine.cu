#include "hip/hip_runtime.h"
#include "sha256.cuh"

extern "C"
__global__ void mine(long* nonces, unsigned char* results, int difficulty, long index, long timestamp, const char* previousHash, const char* merkleRoot)
{
    // Calculate the unique index based on the block and thread dimensions
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Retrieve the nonce that this thread is responsible for
    long nonce = nonces[idx];

    // Buffer to store the concatenated input string (nonce, index, timestamp, previousHash, merkleRoot)
    char input[256];
    int len = 0;

    // Concatenate all the input values into the buffer (index, timestamp, previousHash, nonce, merkleRoot)
    len += sprintf(input + len, "%ld", index); // Index
    len += sprintf(input + len, "%ld", timestamp); // Timestamp
    len += sprintf(input + len, "%s", previousHash); // Previous Hash
    len += sprintf(input + len, "%ld", nonce); // Nonce
    len += sprintf(input + len, "%s", merkleRoot); // Merkle Root

    // SHA256 hash output (32 bytes)
    unsigned char hash[32];

    // Compute the hash
    sha256_kernel((unsigned char*)input, len, hash);  // sha256_kernel is a function in cuSHA

    // Variable to count the number of leading zero bits in the hash
    int leadingZeros = 0;

    // Loop through the bytes of the hash
    for (int i = 0; i < 32; ++i)
    {
        // Loop through each bit of the current byte (8 bits per byte)
        for (int b = 7; b >= 0; --b)
        {
            // Check if the bit is 1
            if ((hash[i] >> b) & 1) {
                // If the bit is 1, stop counting leading zeros
                i = 32; // Exit the loop early
                break;
            }
            
            // If the bit is 0, increment the leading zero count
            leadingZeros++;

            // If the leading zeros count has reached the difficulty requirement, set the result to 1 (valid)
            if (leadingZeros >= difficulty) {
                results[idx] = 1; // Mark the result as valid (1)
                return; // Exit the kernel
            }
        }
    }

    // If we exit the loop and don't find enough leading zeros, mark the result as invalid (0)
    results[idx] = 0;
}